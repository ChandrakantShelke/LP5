#include "hip/hip_runtime.h"
%%writefile matrix_mult.cu
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdbool.h> // Enables use of bool type (true/false).

//Prints a matrix in row-major order with a name header for clarity.
void displayMatrix(int* mat, int rows, int cols, const char* name) {
    printf("\n%s:\n", name);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%4d ", mat[i * cols + j]);
        }
        printf("\n");
    }
}

//!nvcc -arch=sm_75 matrix_mult.cu -o matrix_mult
//!./matrix_mult
//


void matrix_mult(int* a, int* b, int* c, int rowsA, int colsA, int colsB) {
    for (int row = 0; row < rowsA; row++) {
        for (int col = 0; col < colsB; col++) {
            int sum = 0;
            for (int i = 0; i < colsA; i++) {
                sum += a[row * colsA + i] * b[i * colsB + col]; //element from row in A. nd col B
            }
            c[row * colsB + col] = sum; //stores result at row, col.
        }
    }
}

__global__ void matrixMul(int* a, int* b, int* c, int rowsA, int colsA, int colsB) {
    //used to calculate thread’s global position in the grid.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    //ensures thread does not go out of bounds.
    if (row < rowsA && col < colsB) {
        for (int i = 0; i < colsA; i++) {
            sum += a[row * colsA + i] * b[i * colsB + col];
        }
        c[row * colsB + col] = sum;
    }
}

//Compares CPU and GPU result matrices for correctness.
bool verifyMatrixResults(int* c_cuda, int* c_normal, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        if (c_cuda[i] != c_normal[i]) {
            return false;
        }
    }
    return true;
}


int main() {
    int rowsA = 4;
    int colsA = 4;
    int rowsB = 4;
    int colsB = 4;

    int *a, *b, *c_cuda, *c_normal; //Host (CPU) matrices.
    int *dev_a, *dev_b, *dev_c; //Device (GPU) matrices.

   //Allocate memory using malloc.
    a = (int*)malloc(rowsA * colsA * sizeof(int));
    b = (int*)malloc(rowsB * colsB * sizeof(int));
    c_cuda = (int*)malloc(rowsA * colsB * sizeof(int));
    c_normal = (int*)malloc(rowsA * colsB * sizeof(int));

    for (int i = 0; i < rowsA * colsA; i++) {
        a[i] = rand() % 10;
    }
    for (int i = 0; i < rowsB * colsB; i++) {
        b[i] = rand() % 10;
    }

    //Allocate GPU Memory
    hipMalloc((void**)&dev_a, rowsA * colsA * sizeof(int));
    hipMalloc((void**)&dev_b, rowsB * colsB * sizeof(int));
    hipMalloc((void**)&dev_c, rowsA * colsB * sizeof(int));

    //Copy Host to Device - Moves A and B to GPU memory.
    hipMemcpy(dev_a, a, rowsA * colsA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, rowsB * colsB * sizeof(int), hipMemcpyHostToDevice);

    clock_t start_cuda = clock();

    dim3 blockSize(16, 16);
    dim3 gridSize((colsB + blockSize.x - 1) / blockSize.x, (rowsA + blockSize.y - 1) / blockSize.y);
    matrixMul<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, rowsA, colsA, colsB);
    
    //Copy GPU Result to Host
	hipMemcpy(c_cuda, dev_c, rowsA * colsB * sizeof(int), hipMemcpyDeviceToHost);

    clock_t end_cuda = clock();
    double cuda_time = (double)(end_cuda - start_cuda) / CLOCKS_PER_SEC;
    printf("Time Taken GPU : %f", cuda_time);

  // Run CPU Matrix Multiplication and Time It
    clock_t start_normal = clock();
    matrix_mult(a, b, c_normal, rowsA, colsA, colsB);
    clock_t end_normal = clock();
    double normal_time = (double)(end_normal - start_normal) / CLOCKS_PER_SEC;
    printf("\nTime Taken CPU : %f ", normal_time);

  // Compare CPU vs GPU Results
    bool match = verifyMatrixResults(c_cuda, c_normal, rowsA, colsB);
    printf("\nOutput Match: %s", match ? "True" : "False");

  //Calculate and Print Speedup
    double speedup = normal_time / cuda_time;
    printf("\nSpeedup Factor: %f\n", speedup);

    // Display matrices (optional for small size)
    displayMatrix(a, rowsA, colsA, "Matrix A");
    displayMatrix(b, rowsB, colsB, "Matrix B");
    displayMatrix(c_cuda, rowsA, colsB, "Result from GPU");
    displayMatrix(c_normal, rowsA, colsB, "Result from CPU");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c_cuda);
    free(c_normal);

    return 0;
}

//CUDA (Compute Unified Device Architecture)
//parallel computing platform and API created by NVIDIA.
//With CUDA, your GPU (Graphics Processing Unit) can also do computation-heavy tasks

//!nvcc -arch=sm_75 matrix_mult.cu -o matrix_mult
//!./matrix_mult
